#include <iostream>
#include <hip/hip_runtime.h>

#define NUM_STREAMS 3

__global__ void kernel(int *data, int stream_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    data[tid] = stream_id;
}

int main()
{
    const int dataSize = 256;
    int *data;
    hipMallocManaged(&data, dataSize * sizeof(int));

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamCreate(&streams[i]);

    hipEvent_t event, event1;
    float timeInMs;

    hipEventCreate(&event1);
    hipEventCreate(&event);

    std::cout << "waiting for event" << std::endl;
    hipEventSynchronize(event);
    hipEventSynchronize(event1);
    std::cout << "event synchronized" << std::endl;

    hipEventElapsedTime(&timeInMs, event1, event);
    std::cout << "time: " << timeInMs << std::endl;

    hipStreamWaitEvent(streams[0], event);
    // Launch kernels on different streams
    for (int i = 0; i < NUM_STREAMS; i++)
    {
        kernel<<<1, dataSize, 0, streams[i]>>>(data, i);
    }
    hipEventRecord(event, streams[1]);

    // Synchronize higher-level streams with lower-level streams
    hipStreamSynchronize(streams[2]);
    hipStreamSynchronize(streams[1]);

    // Print the results
    for (int i = 0; i < dataSize; i++)
    {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(data);
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamDestroy(streams[i]);

    return 0;
}
