#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#define DATASIZE = 32000000
#define REPEATTIMES = 7

#define cudaCheckError()                                             \
    {                                                                \
        hipError_t e = hipGetLastError();                          \
        if (e != hipSuccess)                                        \
        {                                                            \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
                   hipGetErrorString(e));                           \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

void memCopyP2P(int *des, int desDevice, int *src, int srcDevice,
                int dataSize, int repeat,
                hipEvent_t start, hipEvent_t stop,
                hipStream_t stream);

void memCopyD2D(int *des, int *src, int dataSize, int repeat,
                hipEvent_t start, hipEvent_t stop,
                hipStream_t stream);

void memCopyD2H();

void memCopyH2D();

void memCopy(int desDevice, int srcDevice, int dataSize, int repeat,
                hipEvent_t start, hipEvent_t stop,
                hipStream_t stream);

int numGpus();

int main(int argc, char **argv)
{

}

void memCopyP2P(int *des, int desDevice, int *src, int srcDevice,
                int dataSize, int repeat,
                hipEvent_t start, hipEvent_t stop,
                hipStream_t stream)
{
    hipEventRecord(start, stream);
    cudaCheckError();
    for (int r = 0; r < repeat; r++)
    {
        hipMemcpyPeerAsync(des, desDevice, src, srcDevice,
                            sizeof(int) * dataSize, stream);
        cudaCheckError();
    }
    hipEventRecord(stop, stream);
    cudaCheckError();
}
