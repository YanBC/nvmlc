#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "memeory.h"

#define cudaCheckError()                                             \
    {                                                                \
        hipError_t e = hipGetLastError();                          \
        if (e != hipSuccess)                                        \
        {                                                            \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
                   hipGetErrorString(e));                           \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

// Global variable to control the execution
volatile bool stopExecution = false;

__global__ void delay(volatile int *flag,
                      unsigned long long timeout_clocks = 10000000)
{
    long long int start_clock, sample_clock;
    start_clock = clock64();

    while (true)
    {
        sample_clock = clock64();

        if (sample_clock - start_clock > timeout_clocks)
        {
            break;
        }
    }
}

// Function to occupy bandwidth between GPUs
int occupyBandwidth(int desDevice, int srcDevice, size_t bandwidthInBytesPerSecond)
{
    // Calculate number of bytes in miliseconds
    size_t bytesPerMs = bandwidthInBytesPerSecond / 1000;
    int *desBuffer, *srcBuffer;

    stopExecution = false;

    // Allocate memory
    if (desDevice < 0 && srcDevice < 0)
    {
        return -2;
    }

    if (desDevice < 0)
    {
        desBuffer = (int *)malloc(bytesPerMs)
    }
    else
    {
        hipSetDevice(desDevice);
        cudaCheckError();
        hipMalloc((void **)&desBuffer, bytesPerMs)
    }
    if (srcDevice < 0)
    {
        srcBuffer = (int *)malloc(bytesPerMs)
    }
    else
    {
        hipSetDevice(srcDevice);
        cudaCheckError();
        hipMalloc((void **)&srcBuffer, bytesPerMs)
    }

    // Create CUDA stream
    hipStream_t stream;
    hipEvent_t start, stop;

    hipStreamCreate(&stream);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the execution loop
    while (!stopExecution)
    {
        // Copy data from srcGPU to desGPU using hipMemcpyPeerAsync
        hipMemcpyPeerAsync(desGPU, 0, srcGPU, 0, numElements * sizeof(float), stream);

        // Synchronize the CUDA stream
        hipStreamSynchronize(stream);
    }

    // Release allocated memory
    hipFree(desGPU);
    hipFree(srcGPU);

    // Destroy CUDA stream
    hipStreamDestroy(stream);
}

int main()
{
    // Example usage
    float *desGPU;
    float *srcGPU;
    size_t bandwidthBytes = 1024 * 1024; // Example bandwidth of 1 MB

    // Start the function to occupy bandwidth
    occupyBandwidth(desGPU, srcGPU, bandwidthBytes);

    return 0;
}
